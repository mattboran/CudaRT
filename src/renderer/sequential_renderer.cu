#include "hip/hip_runtime.h"
#include "renderer.h"

#include <algorithm>
#include <iterator>
#include <string.h>

using namespace geom;

SequentialRenderer::SequentialRenderer(Scene* _scenePtr, int _width, int _height, int _samples, bool _useBVH) :
  Renderer(_scenePtr, _width, _height, _samples, _useBVH)
{
    int numTriangles = p_scene->getNumTriangles();
    int numLights = p_scene->getNumLights();
    Triangle* p_triangles = p_scene->getTriPtr();
    Triangle* p_lights = p_scene->getLightsPtr();

    size_t trianglesBytes = sizeof(Triangle) * numTriangles;
    size_t lightsBytes = sizeof(Triangle) * numLights;
    size_t trianglesDataBytes = sizeof(TrianglesData) + trianglesBytes;
    size_t lightsDataBytes = sizeof(LightsData) + lightsBytes;
    h_trianglesData = (TrianglesData*)malloc(trianglesDataBytes);
    h_lightsData = (LightsData*)malloc(lightsDataBytes);
    h_imgBytesPtr = new uchar4[width * height];
    h_imgVectorPtr = new Vector3Df[width * height];

    createTrianglesData(h_trianglesData, p_triangles);
    createLightsData(h_lightsData, p_lights);
    createSettingsData(&h_settingsData);
}

SequentialRenderer::~SequentialRenderer() {
    free(h_trianglesData);
    free(h_lightsData);
    delete[] h_imgBytesPtr;
    delete[] h_imgVectorPtr;
}

__host__ void SequentialRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	samplesRendered++;
	Camera* p_camera = p_scene->getCameraPtr();
	Sampler* p_sampler = new Sampler;
    for (unsigned x = 0; x < width; x++) {
        for (unsigned y = 0; y < height; y++) {
            int idx = y * width + x;
            h_imgVectorPtr[idx] += samplePixel(x, y, p_camera, h_trianglesData, h_lightsData, p_sampler);
            p_img[idx] = vector3ToUchar4(h_imgVectorPtr[idx]/samplesRendered);
        }
    }
	delete p_sampler;
}

__host__ void SequentialRenderer::copyImageBytes() {
	int pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	memcpy(h_imgPtr, h_imgBytesPtr, imgBytes);
	for (unsigned i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}
