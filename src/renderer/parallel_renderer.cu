#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "renderer.h"
#include "cuda_error_check.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define BLOCK_WIDTH 16u

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		TrianglesData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, int _width, int _height, int _samples, bool _useBVH) :
	Renderer(_scenePtr, _width, _height, _samples, _useBVH) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	int pixels = width * height;
	unsigned int numTris = p_scene->getNumTriangles();
	unsigned int numBvhNodes = p_scene->getNumBvhNodes();
	unsigned int numLights = p_scene->getNumLights();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock * gridBlocks;

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_bvhPtr = NULL;
	d_lightsPtr = NULL;
	d_trianglesData = NULL;
	d_lightsData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(Vector3Df) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_bvhPtr, bvhBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsPtr, lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_trianglesData, sizeof(TrianglesData) + trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsData, sizeof(LightsData) + lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	createSettingsData(&d_settingsData);
	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_bvhPtr);
	hipFree(d_lightsPtr);
	hipFree(d_trianglesData);
	hipFree(d_lightsData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	Scene* scenePtr = getScenePtr();
	int numTris = scenePtr->getNumTriangles();
	int numLights = scenePtr->getNumLights();
	int numBvhNodes = scenePtr->getNumBvhNodes();
	float lightsSurfaceArea = scenePtr->getLightsSurfaceArea();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t trianglesDataBytes = sizeof(TrianglesData) + trianglesBytes;

	Camera* h_camPtr = scenePtr->getCameraPtr();
	Triangle* h_triPtr = scenePtr->getTriPtr();
	LinearBVHNode* h_bvhPtr = scenePtr->getBvhPtr();
	Triangle* h_lightsPtr = scenePtr->getLightsPtr();
	TrianglesData* h_trianglesData = (TrianglesData*)malloc(sizeof(TrianglesData) + trianglesBytes + bvhBytes);
	LightsData* h_lightsData = (LightsData*)malloc(sizeof(LightsData) + lightsBytes);

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_bvhPtr, h_bvhPtr, bvhBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsPtr, h_lightsPtr, sizeof(Triangle) * numLights, hipMemcpyHostToDevice));

	createTrianglesData(h_trianglesData, d_triPtr, d_bvhPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_trianglesData, h_trianglesData, trianglesDataBytes, hipMemcpyHostToDevice));

	createLightsData(h_lightsData, d_lightsPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsData, h_lightsData, sizeof(LightsData) + lightsBytes, hipMemcpyHostToDevice));

	free(h_trianglesData);
	free(h_lightsData);
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<grid, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	renderKernel<<<grid, block, 0>>>(d_settingsData,
			d_imgVectorPtr,
			p_img,
			d_camPtr,
			d_trianglesData,
			d_lightsData,
			d_curandStatePtr,
			samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes() {
	int pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, d_imgBytesPtr, imgBytes, hipMemcpyDeviceToHost));
	for (unsigned i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		TrianglesData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = y * settings.width + x;
	hiprandState* p_threadCurand = &p_curandState[idx];
	Sampler sampler(p_threadCurand);
	Vector3Df color = samplePixel(x, y, p_camera, p_tris, p_lights, &sampler);
	p_imgBuffer[idx] += color;
	p_outImg[idx] = vector3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
