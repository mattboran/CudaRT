#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "renderer.h"
#include "cuda_error_check.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define USE_SHARED_MEMORY
#define BLOCK_WIDTH 16u

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		TrianglesData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, pixels_t _width, pixels_t _height, int _samples) :
	Renderer(_scenePtr, _width, _height, _samples) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	pixels_t pixels = width * height;
	uint numTris = p_scene->getNumTriangles();
	uint numMaterials = p_scene->getNumMaterials();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numLights = p_scene->getNumLights();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock * gridBlocks;

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_bvhPtr = NULL;
	d_materials = NULL;
	d_lightsPtr = NULL;
	d_trianglesData = NULL;
	d_lightsData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(Vector3Df) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_materials, materialsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_bvhPtr, bvhBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsPtr, lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_trianglesData, sizeof(TrianglesData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsData, sizeof(LightsData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	createSettingsData(&d_settingsData);
	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_bvhPtr);
	hipFree(d_materials);
	hipFree(d_lightsPtr);
	hipFree(d_trianglesData);
	hipFree(d_lightsData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	unsigned int numTris = p_scene->getNumTriangles();
	unsigned int numLights = p_scene->getNumLights();
	unsigned int numBvhNodes = p_scene->getNumBvhNodes();
	unsigned int numMaterials = p_scene->getNumMaterials();
	float lightsSurfaceArea = p_scene->getLightsSurfaceArea();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;

	Camera* h_camPtr = p_scene->getCameraPtr();
	Triangle* h_triPtr = p_scene->getTriPtr();
	LinearBVHNode* h_bvhPtr = p_scene->getBvhPtr();
	Triangle* h_lightsPtr = p_scene->getLightsPtr();
	Material* h_materialsPtr = p_scene->getMaterialsPtr();
	TrianglesData* h_trianglesData = (TrianglesData*)malloc(sizeof(TrianglesData));
	LightsData* h_lightsData = (LightsData*)malloc(sizeof(LightsData));

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_materials, h_materialsPtr, materialsBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_bvhPtr, h_bvhPtr, bvhBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsPtr, h_lightsPtr, lightsBytes, hipMemcpyHostToDevice));

	createTrianglesData(h_trianglesData, d_triPtr, d_bvhPtr, d_materials);
	CUDA_CHECK_RETURN(hipMemcpy(d_trianglesData, h_trianglesData, sizeof(TrianglesData), hipMemcpyHostToDevice));

	createLightsData(h_lightsData, d_lightsPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsData, h_lightsData, sizeof(LightsData), hipMemcpyHostToDevice));

	free(h_trianglesData);
	free(h_lightsData);
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<grid, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	size_t sharedMemory = sizeof(Material) * p_scene->getNumMaterials();
	renderKernel<<<grid, block, sharedMemory>>>(d_settingsData,
			d_imgVectorPtr,
			p_img,
			d_camPtr,
			d_trianglesData,
			d_lightsData,
			d_curandStatePtr,
			samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes(uchar4* p_img) {
	pixels_t pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, p_img, imgBytes, hipMemcpyDeviceToHost));
	for (unsigned i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		TrianglesData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber) {

#ifdef USE_SHARED_MEMORY
	unsigned int numMaterials = p_tris->numMaterials;
	extern __shared__ Material d_materials[];
	if (threadIdx.x + threadIdx.y == 0) {
		for (int i = 0; i < numMaterials; i++) {
			d_materials[i] = p_tris->p_materials[i];
		}
	}
	__syncthreads();
#else
	Material* d_materials = p_tris->p_materials;
#endif
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint idx = y * settings.width + x;
	hiprandState* p_threadCurand = &p_curandState[idx];
	Sampler sampler(p_threadCurand);
	Vector3Df color = samplePixel(x, y, p_camera, p_tris, p_lights, d_materials, &sampler);
	p_imgBuffer[idx] += color;
	p_outImg[idx] = vector3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
