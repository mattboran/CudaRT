#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "renderer.h"
#include "cuda_error_check.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define USE_SHARED_MEMORY
#define BLOCK_WIDTH 16u

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, pixels_t _width, pixels_t _height, uint _samples) :
	Renderer(_scenePtr, _width, _height, _samples) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	pixels_t pixels = width * height;
	uint numTris = p_scene->getNumTriangles();
	uint numMaterials = p_scene->getNumMaterials();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numLights = p_scene->getNumLights();
	uint numTextures = p_scene->getNumTextures();
	pixels_t totalTexturePixels = p_scene->getTotalTexturePixels();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock * gridBlocks;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_bvhPtr = NULL;
	d_materials = NULL;
	d_textureOffsets = NULL;
	d_cudaTexObjects = NULL;
	d_lightsPtr = NULL;
	d_sceneData = NULL;
	d_lightsData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(Vector3Df) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_materials, materialsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_bvhPtr, bvhBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsPtr, lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sceneData, sizeof(SceneData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsData, sizeof(LightsData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_cudaTexObjects, textureObjectBytes));

	createSettingsData(&d_settingsData);
	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_bvhPtr);
	hipFree(d_materials);
	hipFree(d_cudaTexObjects);
	hipFree(d_lightsPtr);
	hipFree(d_sceneData);
	hipFree(d_lightsData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	uint numTris = p_scene->getNumTriangles();
	uint numLights = p_scene->getNumLights();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numMaterials = p_scene->getNumMaterials();
	uint numTextures = p_scene->getNumTextures();
	pixels_t numTotalTexturePixels = p_scene->getTotalTexturePixels();

	float lightsSurfaceArea = p_scene->getLightsSurfaceArea();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	Camera* h_camPtr = p_scene->getCameraPtr();
	Triangle* h_triPtr = p_scene->getTriPtr();
	LinearBVHNode* h_bvhPtr = p_scene->getBvhPtr();
	Triangle* h_lightsPtr = p_scene->getLightsPtr();
	Material* h_materialsPtr = p_scene->getMaterialsPtr();
	SceneData* h_sceneData = (SceneData*)malloc(sizeof(SceneData));
	LightsData* h_lightsData = (LightsData*)malloc(sizeof(LightsData));
	Vector3Df* h_textureData = p_scene->getTexturePtr();

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_materials, h_materialsPtr, materialsBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_bvhPtr, h_bvhPtr, bvhBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsPtr, h_lightsPtr, lightsBytes, hipMemcpyHostToDevice));

	hipTextureObject_t* h_textureObjects = createTextureObjects();
	CUDA_CHECK_RETURN(hipMemcpy(d_cudaTexObjects, h_textureObjects, textureObjectBytes, hipMemcpyHostToDevice));
	h_sceneData->p_cudaTexObjects = d_cudaTexObjects;

	createSceneData(h_sceneData, d_triPtr, d_bvhPtr, d_materials, NULL, NULL, NULL);
	CUDA_CHECK_RETURN(hipMemcpy(d_sceneData, h_sceneData, sizeof(SceneData), hipMemcpyHostToDevice));

	createLightsData(h_lightsData, d_lightsPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsData, h_lightsData, sizeof(LightsData), hipMemcpyHostToDevice));

	free(h_sceneData);
	free(h_lightsData);
}

__host__ hipTextureObject_t* ParallelRenderer::createTextureObjects() {
	uint numTextures = p_scene->getNumTextures();
	hipTextureObject_t* p_cudaTexObjects = new hipTextureObject_t[numTextures + TEXTURES_OFFSET];
	//
	// BVH
	//
	LinearBVHNode* h_bvh = p_scene->getBvhPtr();
	size_t numBvhNodes = p_scene->getNumBvhNodes();
	// Copy min and max
	{
		size_t size = numBvhNodes * 2 * sizeof(float4);
		float4* h_buffer = new float4[numBvhNodes * 2];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[2*i] = make_float4(h_bvh->min);
			h_buffer[2*i + 1] = make_float4(h_bvh->max);
			h_bvh++;
		}
		float4* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_BOUNDS_OFFSET] = currentTexObject;
		delete h_buffer;
	}
	// Copy indexes, numTriangles, and axis
	{
		size_t size = numBvhNodes * sizeof(int2);
		int2* h_buffer = new int2[numBvhNodes];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[i].x = h_bvh->secondChildOffset;
			//
			int32_t yValue = ((int32_t)(h_bvh->numTriangles) < 16) | ((int32_t)(h_bvh->axis));
			h_buffer[i].y = yValue;
			h_bvh++;
		}
		int2* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindSigned);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_BOUNDS_OFFSET] = currentTexObject;
		delete h_buffer;
	}

	//
	// Actual Textures
	//
	pixels_t* h_textureDimensions = p_scene->getTextureDimensionsPtr();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	for (uint i = 0; i < numTextures; i++) {
		Vector3Df* p_currentTextureData = p_scene->getTexturePtr(i);
		pixels_t width = h_textureDimensions[2*i];
		pixels_t height = h_textureDimensions[2*i + 1];
		pixels_t numPixels = width * height;
		size_t size = numPixels * sizeof(float4);
		float4* p_currentTextureFormattedData = new float4[numPixels];
		for (pixels_t j = 0; j < numPixels; j++) {
			p_currentTextureFormattedData[j] = make_float4(p_currentTextureData[j]);
		}
		hipArray* cuArray = NULL;
		CUDA_CHECK_RETURN(hipMallocArray(&cuArray, &channelDesc, width, height));
		CUDA_CHECK_RETURN(hipMemcpyToArray(cuArray,
											0,
											0,
											p_currentTextureFormattedData,
											size,
											hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[i + TEXTURES_OFFSET] = currentTexObject;
		delete p_currentTextureFormattedData;
	}
	return p_cudaTexObjects;
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<grid, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	size_t sharedMemory = sizeof(Material) * p_scene->getNumMaterials();
	renderKernel<<<grid, block, sharedMemory>>>(d_settingsData,
			d_imgVectorPtr,
			p_img,
			d_camPtr,
			d_sceneData,
			d_lightsData,
			d_curandStatePtr,
			samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes(uchar4* p_img) {
	pixels_t pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, p_img, imgBytes, hipMemcpyDeviceToHost));
	for (uint i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	uint idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber) {

#ifdef USE_SHARED_MEMORY
	unsigned int numMaterials = p_tris->numMaterials;
	extern __shared__ Material d_materials[];
	if (threadIdx.x + threadIdx.y == 0) {
		for (int i = 0; i < numMaterials; i++) {
			d_materials[i] = p_tris->p_materials[i];
		}
	}
	__syncthreads();
#else
	Material* d_materials = p_tris->p_materials;
#endif
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint idx = y * settings.width + x;
	hiprandState* p_threadCurand = &p_curandState[idx];
	Sampler sampler(p_threadCurand);
	Vector3Df color = samplePixel(x, y, p_camera, p_tris, p_lights, d_materials, &sampler);
	p_imgBuffer[idx] += color;
	p_outImg[idx] = vector3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
