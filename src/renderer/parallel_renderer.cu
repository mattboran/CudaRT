#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "cuda_error_check.h"
#include "renderer.h"
#include "scene.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define BLOCK_WIDTH 16u

__constant__ float3 c_materialFloats[MAX_MATERIALS * MATERIALS_FLOAT_COMPONENTS];
__constant__ int2 c_materialIndices[MAX_MATERIALS];
__constant__ pixels_t c_width;
__constant__ float c_lightsSurfaceArea;
__constant__ uint c_numLights;

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(float3* p_imgBuffer,
							 uchar4* p_outImg,
							 Camera camera,
							 SceneData* p_sceneData,
							 uint* p_lightsIndices,
							 hiprandState *p_curandState,
							 int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, pixels_t _width, pixels_t _height, uint _samples) :
	Renderer(_scenePtr, _width, _height, _samples) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	pixels_t pixels = width * height;
	uint numTris = p_scene->getNumTriangles();
	uint numMaterials = p_scene->getNumMaterials();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numLights = p_scene->getNumLights();
	uint numTextures = p_scene->getNumTextures();
	pixels_t totalTexturePixels = p_scene->getTotalTexturePixels();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_cudaTexObjects = NULL;
	d_lightsIndices = NULL;
	d_sceneData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(float3) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsIndices, sizeof(uint) * numLights));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sceneData, sizeof(SceneData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_cudaTexObjects, textureObjectBytes));

	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_cudaTexObjects);
	hipFree(d_lightsIndices);
	hipFree(d_sceneData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	uint numTris = p_scene->getNumTriangles();
	uint numLights = p_scene->getNumLights();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numMaterials = p_scene->getNumMaterials();
	uint numTextures = p_scene->getNumTextures();
	pixels_t numTotalTexturePixels = p_scene->getTotalTexturePixels();
	float lightsSurfaceArea = p_scene->getLightsSurfaceArea();

	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t lightsIndicesBytes = sizeof(uint) * numLights;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	Camera* h_camPtr = p_scene->getCameraPtr();
	Triangle* h_triPtr = p_scene->getTriPtr();
	Material* h_materialsPtr = p_scene->getMaterialsPtr();
	SceneData* h_sceneData = (SceneData*)malloc(sizeof(SceneData));
	uint* h_lightsIndices = p_scene->getLightsIndicesPtr();

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));

	hipTextureObject_t* h_textureObjects = createTextureObjects();
	CUDA_CHECK_RETURN(hipMemcpy(d_cudaTexObjects, h_textureObjects, textureObjectBytes, hipMemcpyHostToDevice));

	h_sceneData->p_triangles = d_triPtr;
	h_sceneData->p_cudaTexObjects = d_cudaTexObjects;
	CUDA_CHECK_RETURN(hipMemcpy(d_sceneData, h_sceneData, sizeof(SceneData), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsIndices, h_lightsIndices, lightsIndicesBytes, hipMemcpyHostToDevice));

	createMaterialsData();

	hipMemcpyToSymbol(HIP_SYMBOL(c_numLights), &numLights, sizeof(uint));
	hipMemcpyToSymbol(HIP_SYMBOL(c_lightsSurfaceArea), &lightsSurfaceArea, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(c_width), &width, sizeof(pixels_t));

	free(h_sceneData);
}

__host__ hipTextureObject_t* ParallelRenderer::createTextureObjects() {
	uint numTextures = p_scene->getNumTextures();
	hipTextureObject_t* p_cudaTexObjects = new hipTextureObject_t[numTextures + TEXTURES_OFFSET];
	//
	// BVH
	//
	LinearBVHNode* h_bvh = p_scene->getBvhPtr();
	size_t numBvhNodes = p_scene->getNumBvhNodes();
	// Copy min and max
	{
		size_t size = numBvhNodes * 2 * sizeof(float4);
		float4* h_buffer = new float4[numBvhNodes * 2];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[2*i] = make_float4(h_bvh[i].min);
			h_buffer[2*i + 1] = make_float4(h_bvh[i].max);
		}
		float4* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_BOUNDS_OFFSET] = currentTexObject;
		delete h_buffer;
	}
	// Copy indexes, numTriangles, and axis
	{
		size_t size = numBvhNodes * sizeof(int2);
		int2* h_buffer = new int2[numBvhNodes];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[i].x = h_bvh->secondChildOffset;
			//
			int32_t yValue = ((int32_t)(h_bvh->numTriangles) << 16) | ((int32_t)(h_bvh->axis));
			h_buffer[i].y = yValue;
			h_bvh++;
		}
		int2* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindSigned);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_INDEX_OFFSET] = currentTexObject;
		delete h_buffer;
	}

	//
	// Actual Textures
	//
	pixels_t* h_textureDimensions = p_scene->getTextureDimensionsPtr();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	for (uint i = 0; i < numTextures; i++) {
		float3* p_currentTextureData = p_scene->getTexturePtr(i);
		pixels_t width = h_textureDimensions[2*i];
		pixels_t height = h_textureDimensions[2*i + 1];
		pixels_t numPixels = width * height;
		size_t size = numPixels * sizeof(float4);
		float4* p_currentTextureFormattedData = new float4[numPixels];
		for (pixels_t j = 0; j < numPixels; j++) {
			p_currentTextureFormattedData[j] = make_float4(p_currentTextureData[j]);
		}
		hipArray* cuArray = NULL;
		CUDA_CHECK_RETURN(hipMallocArray(&cuArray, &channelDesc, width, height));
		CUDA_CHECK_RETURN(hipMemcpyToArray(cuArray,
											0,
											0,
											p_currentTextureFormattedData,
											size,
											hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[i + TEXTURES_OFFSET] = currentTexObject;
		delete p_currentTextureFormattedData;
	}
	return p_cudaTexObjects;
}

__host__ void ParallelRenderer::createMaterialsData() {
	Material* p_materials = p_scene->getMaterialsPtr();
	uint numMaterials = p_scene->getNumMaterials();
	float3* p_floatBuffer = new float3[MAX_MATERIALS * MATERIALS_FLOAT_COMPONENTS];
	int2* p_intBuffer = new int2[MAX_MATERIALS];
	float3* p_currentFloat = p_floatBuffer;
	int2* p_currentIndex = p_intBuffer;
	for (uint i = 0; i < numMaterials; i++) {
		*p_currentFloat++ = p_materials[i].kd;
		*p_currentFloat++ = p_materials[i].ka;
		*p_currentFloat++ = p_materials[i].ks;
		*p_currentFloat++ = make_float3(p_materials[i].ns,
										p_materials[i].ni,
										p_materials[i].diffuseCoefficient);
		*p_currentIndex++ = make_int2((int32_t)p_materials[i].bsdf,
									  (int32_t)p_materials[i].texKdIdx);

	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_materialFloats),
					   p_floatBuffer,
					   numMaterials * MATERIALS_FLOAT_COMPONENTS * sizeof(float3));
	hipMemcpyToSymbol(HIP_SYMBOL(c_materialIndices),
					   p_intBuffer,
					   numMaterials * sizeof(int2));

	delete p_floatBuffer;
	delete p_intBuffer;
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<1, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	Camera camera = *p_scene->getCameraPtr();
	size_t sharedBytes = sizeof(Sampler) * BLOCK_WIDTH * BLOCK_WIDTH;
	renderKernel<<<grid, block, sharedBytes>>>(d_imgVectorPtr,
												p_img,
												camera,
												d_sceneData,
												d_lightsIndices,
												d_curandStatePtr,
												samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes(uchar4* p_img) {
	pixels_t pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, p_img, imgBytes, hipMemcpyDeviceToHost));
	for (uint i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	uint idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(float3* p_imgBuffer,
							uchar4* p_outImg,
							Camera camera,
							SceneData* p_sceneData,
							uint* p_lightsIndices,
							hiprandState *p_curandState,
							int sampleNumber) {
	extern __shared__ Sampler p_samplers[];
	uint x = (blockIdx.x * blockDim.x + threadIdx.x);
	uint y = (blockIdx.y * blockDim.y + threadIdx.y);
	uint blockOnlyIdx = threadIdx.x * blockDim.x + threadIdx.y;
	uint idx = y * c_width + x;
	p_samplers[blockOnlyIdx] = Sampler(&p_curandState[blockOnlyIdx]);
	float3 color = samplePixel(x, y,
								  camera,
								  p_sceneData,
								  p_lightsIndices,
								  c_numLights,
								  c_lightsSurfaceArea,
								  &p_samplers[blockOnlyIdx],
								  c_materialFloats,
								  c_materialIndices);
	p_imgBuffer[idx] = p_imgBuffer[idx] + color;
	p_outImg[idx] = float3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
