#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "renderer.h"
#include "cuda_error_check.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define BLOCK_WIDTH 16u

__constant__ float3 c_materialFloats[MAX_MATERIALS * MATERIALS_FLOAT_COMPONENTS];
__constant__ int2 c_materialIndices[MAX_MATERIALS];

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_sceneData,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, pixels_t _width, pixels_t _height, uint _samples) :
	Renderer(_scenePtr, _width, _height, _samples) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	pixels_t pixels = width * height;
	uint numTris = p_scene->getNumTriangles();
	uint numMaterials = p_scene->getNumMaterials();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numLights = p_scene->getNumLights();
	uint numTextures = p_scene->getNumTextures();
	pixels_t totalTexturePixels = p_scene->getTotalTexturePixels();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_cudaTexObjects = NULL;
	d_lightsPtr = NULL;
	d_sceneData = NULL;
	d_lightsData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(Vector3Df) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsPtr, lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sceneData, sizeof(SceneData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsData, sizeof(LightsData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_cudaTexObjects, textureObjectBytes));

	createSettingsData(&d_settingsData);
	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_cudaTexObjects);
	hipFree(d_lightsPtr);
	hipFree(d_sceneData);
	hipFree(d_lightsData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	uint numTris = p_scene->getNumTriangles();
	uint numLights = p_scene->getNumLights();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numMaterials = p_scene->getNumMaterials();
	uint numTextures = p_scene->getNumTextures();
	pixels_t numTotalTexturePixels = p_scene->getTotalTexturePixels();

	float lightsSurfaceArea = p_scene->getLightsSurfaceArea();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t textureObjectBytes = sizeof(hipTextureObject_t) * (numTextures + TEXTURES_OFFSET);

	Camera* h_camPtr = p_scene->getCameraPtr();
	Triangle* h_triPtr = p_scene->getTriPtr();
	Triangle* h_lightsPtr = p_scene->getLightsPtr();
	Material* h_materialsPtr = p_scene->getMaterialsPtr();
	SceneData* h_sceneData = (SceneData*)malloc(sizeof(SceneData));
	LightsData* h_lightsData = (LightsData*)malloc(sizeof(LightsData));
	Vector3Df* h_textureData = p_scene->getTexturePtr();

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsPtr, h_lightsPtr, lightsBytes, hipMemcpyHostToDevice));

	hipTextureObject_t* h_textureObjects = createTextureObjects();
	CUDA_CHECK_RETURN(hipMemcpy(d_cudaTexObjects, h_textureObjects, textureObjectBytes, hipMemcpyHostToDevice));
	h_sceneData->p_cudaTexObjects = d_cudaTexObjects;

	createSceneData(h_sceneData, d_triPtr, NULL, NULL, NULL, NULL);
	CUDA_CHECK_RETURN(hipMemcpy(d_sceneData, h_sceneData, sizeof(SceneData), hipMemcpyHostToDevice));

	createLightsData(h_lightsData, d_lightsPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsData, h_lightsData, sizeof(LightsData), hipMemcpyHostToDevice));

	createMaterialsData(NULL, NULL);
	free(h_sceneData);
	free(h_lightsData);
}

__host__ hipTextureObject_t* ParallelRenderer::createTextureObjects() {
	uint numTextures = p_scene->getNumTextures();
	hipTextureObject_t* p_cudaTexObjects = new hipTextureObject_t[numTextures + TEXTURES_OFFSET];
	//
	// BVH
	//
	LinearBVHNode* h_bvh = p_scene->getBvhPtr();
	size_t numBvhNodes = p_scene->getNumBvhNodes();
	// Copy min and max
	{
		size_t size = numBvhNodes * 2 * sizeof(float4);
		float4* h_buffer = new float4[numBvhNodes * 2];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[2*i] = make_float4(h_bvh[i].min);
			h_buffer[2*i + 1] = make_float4(h_bvh[i].max);
		}
		float4* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_BOUNDS_OFFSET] = currentTexObject;
		delete h_buffer;
	}
	// Copy indexes, numTriangles, and axis
	{
		size_t size = numBvhNodes * sizeof(int2);
		int2* h_buffer = new int2[numBvhNodes];
		for (uint i = 0; i < numBvhNodes; i++) {
			h_buffer[i].x = h_bvh->secondChildOffset;
			//
			int32_t yValue = ((int32_t)(h_bvh->numTriangles) << 16) | ((int32_t)(h_bvh->axis));
			h_buffer[i].y = yValue;
			h_bvh++;
		}
		int2* d_buffer = NULL;
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_buffer, size));
		CUDA_CHECK_RETURN(hipMemcpy(d_buffer, h_buffer, size, hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = d_buffer;
		resDesc.res.linear.desc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindSigned);
		resDesc.res.linear.sizeInBytes = size;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[BVH_INDEX_OFFSET] = currentTexObject;
		delete h_buffer;
	}

	//
	// Actual Textures
	//
	pixels_t* h_textureDimensions = p_scene->getTextureDimensionsPtr();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	for (uint i = 0; i < numTextures; i++) {
		Vector3Df* p_currentTextureData = p_scene->getTexturePtr(i);
		pixels_t width = h_textureDimensions[2*i];
		pixels_t height = h_textureDimensions[2*i + 1];
		pixels_t numPixels = width * height;
		size_t size = numPixels * sizeof(float4);
		float4* p_currentTextureFormattedData = new float4[numPixels];
		for (pixels_t j = 0; j < numPixels; j++) {
			p_currentTextureFormattedData[j] = make_float4(p_currentTextureData[j]);
		}
		hipArray* cuArray = NULL;
		CUDA_CHECK_RETURN(hipMallocArray(&cuArray, &channelDesc, width, height));
		CUDA_CHECK_RETURN(hipMemcpyToArray(cuArray,
											0,
											0,
											p_currentTextureFormattedData,
											size,
											hipMemcpyHostToDevice));

		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.filterMode = hipFilterModeLinear;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		hipTextureObject_t currentTexObject = 0;
		hipCreateTextureObject(&currentTexObject,
								&resDesc,
								&texDesc,
								NULL);
		p_cudaTexObjects[i + TEXTURES_OFFSET] = currentTexObject;
		delete p_currentTextureFormattedData;
	}
	return p_cudaTexObjects;
}

__host__ void ParallelRenderer::createMaterialsData(float3* matFloats, int2* matIndices) {
	Material* p_materials = p_scene->getMaterialsPtr();
	uint numMaterials = p_scene->getNumMaterials();
	float3* p_floatBuffer = new float3[MAX_MATERIALS * MATERIALS_FLOAT_COMPONENTS];
	int2* p_intBuffer = new int2[MAX_MATERIALS];
	float3* p_currentFloat = p_floatBuffer;
	int2* p_currentIndex = p_intBuffer;
	for (uint i = 0; i < numMaterials; i++) {
		*p_currentFloat++ = make_float3(p_materials[i].kd);
		*p_currentFloat++ = make_float3(p_materials[i].ka);
		*p_currentFloat++ = make_float3(p_materials[i].ks);
		*p_currentFloat++ = make_float3(p_materials[i].ns,
										p_materials[i].ni,
										p_materials[i].diffuseCoefficient);
		*p_currentIndex++ = make_int2((int32_t)p_materials[i].bsdf,
									  (int32_t)p_materials[i].texKdIdx);

	}
	hipMemcpyToSymbol(HIP_SYMBOL(c_materialFloats),
					   p_floatBuffer,
					   numMaterials * MATERIALS_FLOAT_COMPONENTS * sizeof(float3));
	hipMemcpyToSymbol(HIP_SYMBOL(c_materialIndices),
					   p_intBuffer,
					   numMaterials * sizeof(int2));

	delete p_floatBuffer;
	delete p_intBuffer;
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<1, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	renderKernel<<<grid, block, 0>>>(d_settingsData,
			d_imgVectorPtr,
			p_img,
			d_camPtr,
			d_sceneData,
			d_lightsData,
			d_curandStatePtr,
			samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes(uchar4* p_img) {
	pixels_t pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, p_img, imgBytes, hipMemcpyDeviceToHost));
	for (uint i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	uint idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_sceneData,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber) {

	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint blockOnlyIdx = threadIdx.x * blockDim.x + threadIdx.y;
	uint idx = y * settings.width + x;
	hiprandState* p_threadCurand = &p_curandState[blockOnlyIdx];
	Sampler sampler(p_threadCurand);
	Vector3Df color = samplePixel(x, y,
								  p_camera,
								  p_sceneData,
								  p_lights,
								  &sampler,
								  c_materialFloats,
								  c_materialIndices);
	p_imgBuffer[idx] += color;
	p_outImg[idx] = vector3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
