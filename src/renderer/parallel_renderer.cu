#include "hip/hip_runtime.h"
/*
 * parallel_renderer.cpp
 *
 *  Created on: Dec 22, 2018
 *      Author: matt
 */

#include "renderer.h"
#include "cuda_error_check.h"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

using std::cout;

#define USE_SHARED_MEMORY
#define BLOCK_WIDTH 16u

// Kernels
__global__ void initializeCurandKernel(hiprandState* p_curandState);
__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber);

__host__ ParallelRenderer::ParallelRenderer(Scene* _scenePtr, pixels_t _width, pixels_t _height, uint _samples) :
	Renderer(_scenePtr, _width, _height, _samples) {
	// CUDA settings
	useCuda = true;
	threadsPerBlock = BLOCK_WIDTH * BLOCK_WIDTH;
	gridBlocks = width / BLOCK_WIDTH * height / BLOCK_WIDTH;

	pixels_t pixels = width * height;
	uint numTris = p_scene->getNumTriangles();
	uint numMaterials = p_scene->getNumMaterials();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numLights = p_scene->getNumLights();
	uint numTextures = p_scene->getNumTextures();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;
	size_t curandBytes = sizeof(hiprandState) * threadsPerBlock * gridBlocks;

	d_imgVectorPtr = NULL;
	d_imgBytesPtr = NULL;
	d_camPtr = NULL;
	d_triPtr = NULL;
	d_bvhPtr = NULL;
	d_materials = NULL;
	dd_textureData = NULL;
	d_textureDimensions = NULL;
	d_lightsPtr = NULL;
	d_sceneData = NULL;
	d_lightsData = NULL;
	d_curandStatePtr = NULL;

	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgVectorPtr, sizeof(Vector3Df) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgBytesPtr, sizeof(uchar4) * pixels));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, trianglesBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_materials, materialsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_bvhPtr, bvhBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsPtr, lightsBytes));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_sceneData, sizeof(SceneData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lightsData, sizeof(LightsData)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_curandStatePtr, curandBytes));

	CUDA_CHECK_RETURN(hipMalloc((void**)&dd_textureData, sizeof(Vector3Df*) * numTextures));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_textureDimensions, sizeof(pixels_t) * numTextures * 2));
	createSettingsData(&d_settingsData);
	copyMemoryToCuda();

	initializeCurand();
}

__host__ ParallelRenderer::~ParallelRenderer() {
	hipFree(d_imgVectorPtr);
	hipFree(d_camPtr);
	hipFree(d_triPtr);
	hipFree(d_bvhPtr);
	hipFree(d_materials);
	hipFree(dd_textureData);
	hipFree(d_textureDimensions);
	hipFree(d_lightsPtr);
	hipFree(d_sceneData);
	hipFree(d_lightsData);
	hipFree(d_curandStatePtr);
}

__host__ void ParallelRenderer::copyMemoryToCuda() {
	uint numTris = p_scene->getNumTriangles();
	uint numLights = p_scene->getNumLights();
	uint numBvhNodes = p_scene->getNumBvhNodes();
	uint numMaterials = p_scene->getNumMaterials();
	uint numTextures = p_scene->getNumTextures();

	float lightsSurfaceArea = p_scene->getLightsSurfaceArea();
	size_t trianglesBytes = sizeof(Triangle) * numTris;
	size_t materialsBytes = sizeof(Material) * numMaterials;
	size_t bvhBytes = sizeof(LinearBVHNode) * numBvhNodes;
	size_t lightsBytes = sizeof(Triangle) * numLights;

	Camera* h_camPtr = p_scene->getCameraPtr();
	Triangle* h_triPtr = p_scene->getTriPtr();
	LinearBVHNode* h_bvhPtr = p_scene->getBvhPtr();
	Triangle* h_lightsPtr = p_scene->getLightsPtr();
	Material* h_materialsPtr = p_scene->getMaterialsPtr();
	SceneData* h_SceneData = (SceneData*)malloc(sizeof(SceneData));
	LightsData* h_lightsData = (LightsData*)malloc(sizeof(LightsData));

	CUDA_CHECK_RETURN(hipMemcpy(d_camPtr, h_camPtr, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_triPtr, h_triPtr, trianglesBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_materials, h_materialsPtr, materialsBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_bvhPtr, h_bvhPtr, bvhBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsPtr, h_lightsPtr, lightsBytes, hipMemcpyHostToDevice));

	for (uint i = 0; i < numTextures; i++) {
		pixels_t* p_texDimensions = p_scene->getTextureDimensionsPtr();
		pixels_t width = p_texDimensions[i * 2];
		pixels_t height = p_texDimensions[i * 2 + 1];
		size_t pixelsBytes = width * height * sizeof(Vector3Df);
		Vector3Df* d_currentTexture;
		Vector3Df* h_currentTexture = p_scene->getTexturePtr(i);
		CUDA_CHECK_RETURN(hipMalloc((void**)&d_currentTexture, pixelsBytes));
		CUDA_CHECK_RETURN(hipMemcpy(&dd_textureData[i], d_currentTexture, sizeof(Vector3Df*), hipMemcpyHostToDevice));
		CUDA_CHECK_RETURN(hipMemcpy(dd_textureData[i], h_currentTexture, pixelsBytes, hipMemcpyHostToDevice));
	}
	createSceneData(h_SceneData, d_triPtr, d_bvhPtr, d_materials, dd_textureData, d_textureDimensions);
	CUDA_CHECK_RETURN(hipMemcpy(d_sceneData, h_SceneData, sizeof(SceneData), hipMemcpyHostToDevice));

	createLightsData(h_lightsData, d_lightsPtr);
	CUDA_CHECK_RETURN(hipMemcpy(d_lightsData, h_lightsData, sizeof(LightsData), hipMemcpyHostToDevice));

	free(h_SceneData);
	free(h_lightsData);
}

__host__ void ParallelRenderer::initializeCurand() {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);

	initializeCurandKernel<<<grid, block, 0>>>(d_curandStatePtr);
}

__host__ void ParallelRenderer::renderOneSamplePerPixel(uchar4* p_img) {
	dim3 block = dim3(BLOCK_WIDTH, BLOCK_WIDTH, 1);
	dim3 grid = dim3(width/BLOCK_WIDTH, height/BLOCK_WIDTH, 1);
	samplesRendered++;
	size_t sharedMemory = sizeof(Material) * p_scene->getNumMaterials();
	renderKernel<<<grid, block, sharedMemory>>>(d_settingsData,
			d_imgVectorPtr,
			p_img,
			d_camPtr,
			d_sceneData,
			d_lightsData,
			d_curandStatePtr,
			samplesRendered);
}

__host__ void ParallelRenderer::copyImageBytes(uchar4* p_img) {
	pixels_t pixels = width * height;
	size_t imgBytes = sizeof(uchar4) * pixels;
	CUDA_CHECK_RETURN(hipMemcpy(h_imgPtr, p_img, imgBytes, hipMemcpyDeviceToHost));
	for (uint i = 0; i < pixels; i++) {
		gammaCorrectPixel(h_imgPtr[i]);
	}
}

__global__ void initializeCurandKernel(hiprandState* p_curandState) {
	uint idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
				+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &p_curandState[idx]);
}

__global__ void renderKernel(SettingsData settings,
		Vector3Df* p_imgBuffer,
		uchar4* p_outImg,
		Camera* p_camera,
		SceneData* p_tris,
		LightsData* p_lights,
		hiprandState *p_curandState,
		int sampleNumber) {

#ifdef USE_SHARED_MEMORY
	unsigned int numMaterials = p_tris->numMaterials;
	extern __shared__ Material d_materials[];
	if (threadIdx.x + threadIdx.y == 0) {
		for (int i = 0; i < numMaterials; i++) {
			d_materials[i] = p_tris->p_materials[i];
		}
	}
	__syncthreads();
#else
	Material* d_materials = p_tris->p_materials;
#endif
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;
	uint idx = y * settings.width + x;
	hiprandState* p_threadCurand = &p_curandState[idx];
	Sampler sampler(p_threadCurand);
	Vector3Df color = samplePixel(x, y, p_camera, p_tris, p_lights, d_materials, &sampler);
	p_imgBuffer[idx] += color;
	p_outImg[idx] = vector3ToUchar4(p_imgBuffer[idx]/(float)sampleNumber);
}
