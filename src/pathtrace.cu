#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : pathtrace.cu
 Author      : Tudor Matei Boran
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA raytracer
 ============================================================================
 */
#include "camera.cuh"
#include "cuda_textures.cuh"
#include "pathtrace.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "cuda_error_check.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
// Random  number generation with CUDA

using namespace geom;

__global__ void debugRenderKernel(Triangle* d_triPtr, int numTriangles, Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height);
__global__ void renderKernel(Triangle* d_triPtr, int numTriangles, Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height);
__device__ float intersectTriangles(Triangle* d_triPtr, int numTriangles, RayHit& hitData, const Ray& ray);
//__host__ void configureTexture(texture_t &triTexture);
//__host__ hipArray* bindTrianglesToTexture(Triangle* triPtr, unsigned numTris, texture_t &triTexture);
__device__ inline Triangle getTriangleFromTexture(unsigned i);

__device__ static bool* d_useTextureMemory = NULL;

texture_t triangleTexture;
hiprandGenerator_t* generator;

Vector3Df* pathtraceWrapper(Scene& scene, int width, int height, int samples, bool &useTexMemory) {
	int pixels = width * height;
	unsigned numTris = scene.getNumTriangles();
	size_t triangleBytes = sizeof(Triangle) * numTris;
	size_t imageBytes = sizeof(Vector3Df) * width * height;

	// Initialize CUDA memory
	Triangle* triPtr = scene.getTriPtr();
	Triangle* d_triPtr = NULL;
	Vector3Df* imgDataPtr = new Vector3Df[pixels];
	Vector3Df* d_imgDataPtr = NULL;
	Camera* camPtr = scene.getCameraPtr();
	Camera* d_camPtr = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_triPtr, triangleBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_triPtr, (void*)triPtr, triangleBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_imgDataPtr, imageBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_imgDataPtr, (void*)imgDataPtr, imageBytes, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_camPtr, (void*)camPtr, sizeof(Camera), hipMemcpyHostToDevice));

	// Bind triangles to texture memory
	hipArray* d_triDataArray = NULL;
	if (useTexMemory && numTris > TEX_ARRAY_MAX) {
		std::cout << "Not using texture memory because we cannot fit " \
				<< numTris << " triangles in 1D hipArray" << std::endl;
		useTexMemory = false;
	}
	if (useTexMemory) {
		std::cout << "Using texture memory!" << std::endl;
		configureTexture(triangleTexture);
		d_triDataArray = bindTrianglesToTexture(triPtr, numTris, triangleTexture);
	}
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_useTextureMemory, sizeof(bool)));
	CUDA_CHECK_RETURN(hipMemset((void*)d_useTextureMemory, (int)useTexMemory, sizeof(bool)));

	// Launch kernel
	dim3 block(blockWidth, blockWidth, 1);
	dim3 grid(width/blockWidth, height/blockWidth, 1);

	for (int s = 0; s < samples; s++)
	{
		renderKernel <<<grid, block>>>(d_triPtr, numTris, d_camPtr, d_imgDataPtr, width, height);
	}

	CUDA_CHECK_RETURN(hipMemcpy((void*)imgDataPtr, (void*)d_imgDataPtr, imageBytes, hipMemcpyDeviceToHost));
	hipFree((void*)d_triPtr);
	hipFree((void*)d_imgDataPtr);
	if (useTexMemory)
		hipFreeArray(d_triDataArray);
	return imgDataPtr;
}

__global__ void renderKernel(Triangle* d_triPtr, int numTriangles, Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height);

__global__ void debugRenderKernel(geom::Triangle* d_triPtr, int numTriangles, Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height) {
	unsigned int i, j;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	j = blockIdx.y*blockDim.y + threadIdx.y;

	Ray camRay = d_camPtr->computeCameraRay(i, j);
	RayHit hitData;
	float t = intersectTriangles(d_triPtr, numTriangles, hitData, camRay);
	Vector3Df light(0.0f, 10.0f, 1.0f);
	if (t < MAX_DISTANCE) {
		Vector3Df hitPt = camRay.pointAlong(t);
		Vector3Df lightDir = normalize(light - hitPt);
		Vector3Df normal = hitData.hitTriPtr->getNormal(hitData);
		d_imgPtr[j * width + i] = Vector3Df(hitData.hitTriPtr->_colorDiffuse * max(dot(lightDir, normal), 0.0f));
	}
}

__device__ float intersectTriangles(geom::Triangle* d_triPtr, int numTriangles, RayHit& hitData, const Ray& ray) {
	float t = MAX_DISTANCE, tprime = MAX_DISTANCE;
	float u, v;
	for (unsigned i = 0; i < numTriangles; i++)
	{
		Triangle tri;
		if (d_useTextureMemory) {
			Triangle tri = getTriangleFromTexture(i);
			tprime = tri.intersect(ray, u, v);
		} else {
			tprime = d_triPtr[i].intersect(ray, u, v);
		}
		if (tprime < t && tprime > 0.f)
		{
			t = tprime;
			hitData.hitTriPtr = &d_triPtr[i];
			hitData.u = u;
			hitData.v = v;
		}
	}
	return t;
}

__device__ inline Triangle getTriangleFromTexture(unsigned i) {
	float4 v1, e1, e2, n1, n2, n3, diff, spec, emit;
	v1 = tex1Dfetch(triangleTexture, i * 9);
	e1 = tex1Dfetch(triangleTexture, i * 9 + 1);
	e2 = tex1Dfetch(triangleTexture, i * 9 + 2);
	n1 = tex1Dfetch(triangleTexture, i * 9 + 3);
	n2 = tex1Dfetch(triangleTexture, i * 9 + 4);
	n3 = tex1Dfetch(triangleTexture, i * 9 + 5);
	diff = tex1Dfetch(triangleTexture, i * 9 + 6);
	spec = tex1Dfetch(triangleTexture, i * 9 + 7);
	emit = tex1Dfetch(triangleTexture, i * 9 + 8);
	return Triangle(v1, e1, e2, n1, n2, n3, diff, spec, emit);
}

