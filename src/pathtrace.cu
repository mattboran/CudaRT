#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : pathtrace.cu
 Author      : Tudor Matei Boran
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA raytracer
 ============================================================================
 */
#include "camera.cuh"
#include "cuda_textures.cuh"
#include "pathtrace.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <cfloat>
#include "cuda_error_check.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h

using namespace geom;
using namespace std;


texture_t triangleTexture;

Vector3Df* pathtraceWrapper(Scene& scene, int width, int height, int samples, int numStreams, bool &useTexMemory, bool useBvh) {
	int pixels = width * height;
	unsigned numTris = scene.getNumTriangles();
	unsigned numBVHNodes = scene.getNumBVHNodes();

	int numGpus;
	hipGetDeviceCount(&numGpus);
	numStreams = numStreams >= numGpus ? numGpus : numStreams;
	cout << "Got " << numGpus << " cuda-capable devices. Creating "<< numStreams << " streams." << endl;

	size_t triangleBytes = sizeof(Triangle) * numTris;
	size_t imageBytes = sizeof(Vector3Df) * width * height;
	size_t bvhBytes = sizeof(CacheFriendlyBVHNode) * numBVHNodes;

	// Initialize CUDA memory
	// Triangles -> d_tris
	Triangle* h_triPtr = scene.getTriPtr();
	Triangle* d_triPtr = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_triPtr, triangleBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_triPtr, (void* )h_triPtr, triangleBytes, hipMemcpyHostToDevice));

	unsigned* h_bvhIndexPtr = scene.getBVHIndexPtr();
	unsigned* d_bvhIndexPtr = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_bvhIndexPtr, sizeof(unsigned) * numBVHNodes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_bvhIndexPtr, (void* )h_bvhIndexPtr, sizeof(unsigned) * numBVHNodes, hipMemcpyHostToDevice));

	// CacheFriendlyBVHNodes -> d_bvh
	CacheFriendlyBVHNode* h_bvh = scene.getSceneCFBVHPtr();
	CacheFriendlyBVHNode* d_bvh = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_bvh, bvhBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_bvh, (void* )h_bvh, bvhBytes, hipMemcpyHostToDevice));

	TrianglesData* h_tris = (TrianglesData*)malloc(sizeof(TrianglesData) + triangleBytes);
	h_tris->numTriangles = numTris;
	h_tris->triPtr = d_triPtr;
	h_tris->numBVHNodes = numBVHNodes;
	h_tris->bvhPtr = d_bvh;
	h_tris->bvhIndexPtr = d_bvhIndexPtr;
	TrianglesData* d_tris = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&d_tris, sizeof(TrianglesData) + triangleBytes + bvhBytes + sizeof(unsigned) * numBVHNodes));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_tris, (void*)h_tris, sizeof(TrianglesData) + triangleBytes, hipMemcpyHostToDevice));

	// Bind triangles to texture memory -- texture memory doesn't quite work
	hipArray* d_triDataArray = NULL;
	if (useTexMemory && numTris > TEX_ARRAY_MAX) {
		std::cout << "Not using texture memory because we cannot fit "
				<< numTris << " triangles in 1D hipArray" << std::endl;
		useTexMemory = false;
	}
	if (useTexMemory) {
		std::cout << "Using texture memory!" << std::endl;
		configureTexture(triangleTexture);
		d_triDataArray = bindTrianglesToTexture(h_triPtr, numTris,
				triangleTexture);
	}

	// Lights -> d_lights
	Triangle* lightsPtr = scene.getLightsPtr();
	Triangle* d_lightTrianglePtr = NULL;
	size_t lightTrianglesBytes = sizeof(Triangle) * scene.getNumLights();
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_lightTrianglePtr, lightTrianglesBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_lightTrianglePtr, (void *)lightsPtr, lightTrianglesBytes, hipMemcpyHostToDevice));

	LightsData* h_lights = (LightsData*)malloc(sizeof(LightsData) + lightTrianglesBytes);
	h_lights->lightsPtr = d_lightTrianglePtr;
	h_lights->numLights = scene.getNumLights();
	h_lights->totalSurfaceArea = scene.getLightsSurfaceArea();
	LightsData* d_lights = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&d_lights, sizeof(LightsData) + lightTrianglesBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_lights, h_lights, sizeof(LightsData) + lightTrianglesBytes, hipMemcpyHostToDevice));

	// Camera
	Camera* camPtr = scene.getCameraPtr();
	Camera* d_camPtr = NULL;
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_camPtr, (void* )camPtr, sizeof(Camera), hipMemcpyHostToDevice));

	// Setup settings -> d_settings
	SettingsData h_settings;
	h_settings.width = width;
	h_settings.height = height;
	h_settings.samples = samples;
	h_settings.useTexMem = useTexMemory;
	h_settings.numStreams = numStreams;
	h_settings.useBvh = useBvh;
	SettingsData* d_settings;
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&d_settings, sizeof(SettingsData)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_settings, &h_settings, sizeof(SettingsData), hipMemcpyHostToDevice));

	// Launch kernels
	const unsigned int threadsPerBlock = blockWidth * blockWidth;
	const unsigned int gridBlocks = width / blockWidth * height / blockWidth;
	dim3 block(blockWidth, blockWidth, 1);
	dim3 grid(width / blockWidth, height / blockWidth, 1);

	hipStream_t streams[numStreams];

	// Image
	Vector3Df* imgDataPtr = new Vector3Df[pixels]();
	Vector3Df* d_imgDataPtr = NULL;
	Vector3Df* d_streamImgDataPtr;
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_imgDataPtr, imageBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_imgDataPtr, (void* )imgDataPtr, imageBytes, hipMemcpyHostToDevice));

	// Setup cuRand kernel and data in streams
	hiprandState* d_curandState;
	int imagePixels = width * height;
	int curandStateSize = threadsPerBlock * gridBlocks;
	size_t curandStateBytes = sizeof(hiprandState) * curandStateSize;
	CUDA_CHECK_RETURN(hipMallocManaged((void**)&d_streamImgDataPtr, imageBytes * numStreams));
	CUDA_CHECK_RETURN(hipMallocManaged((void** )&d_curandState, curandStateBytes * numStreams));
	for (int s = 0; s < numStreams; s++) {
		CUDA_CHECK_RETURN(hipSetDevice(s));
		CUDA_CHECK_RETURN(hipStreamCreate(&streams[s]));
		hiprandState* d_curandStatePtr = &d_curandState[s * curandStateSize];
		setupCurandKernel<<<grid, block, 0, streams[s]>>>(d_curandStatePtr, s);
		CUDA_CHECK_RETURN(hipMemcpy((void* )&d_streamImgDataPtr[s * imagePixels], (void* )imgDataPtr, imageBytes, hipMemcpyHostToDevice));
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	unsigned streamId = 0;
	int incrementEvery = samples/numStreams;
	CUDA_CHECK_RETURN(hipSetDevice(0));
	for (int s = 1; s <= samples; s++) {
		if (s % incrementEvery == 0) {
			CUDA_CHECK_RETURN(hipSetDevice(streamId++));
		}
		streamId = streamId % numStreams;
		Vector3Df* streamImgData = &d_streamImgDataPtr[streamId * imagePixels];
		hiprandState* d_curandStatePtr = &d_curandState[curandStateSize * streamId];
		renderKernel<<<grid, block, 0, streams[streamId]>>>(d_tris, d_camPtr, streamImgData, d_lights, d_settings, d_curandStatePtr, streamId);
	}

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(hipSetDevice(0));
	// Combine the different streams into a single image
	averageSamplesAndGammaCorrectKernel<<<grid, block>>>(d_streamImgDataPtr, d_imgDataPtr, d_settings);

	CUDA_CHECK_RETURN(hipMemcpy((void* )imgDataPtr, (void* )d_imgDataPtr, imageBytes, hipMemcpyDeviceToHost));

	// Clean up the streams
	for (int s = 0; s < numStreams; s++) {
		CUDA_CHECK_RETURN(hipStreamDestroy(streams[s]));
	}

	// Clean up host memory
	free(h_lights);
	free(h_tris);
	// Clean up device memory
	hipFree((void*) d_tris);
	hipFree((void*) d_triPtr);
	hipFree((void*) d_bvh);
	hipFree((void*) d_lights);
	hipFree((void*) d_lightTrianglePtr);
	hipFree((void*) d_settings);
	hipFree((void*) d_curandState);
	hipFree((void*) d_imgDataPtr);
	hipFree((void*) d_streamImgDataPtr);
	if (useTexMemory)
		hipFreeArray(d_triDataArray);
	return imgDataPtr;
}

__global__ void renderKernel(TrianglesData* d_tris,
							Camera* d_camPtr,
							Vector3Df* d_imgPtr,
							LightsData* d_lights,
							SettingsData* d_settings,
							hiprandState *randState,
							int streamId) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	bool useBvh = d_settings->useBvh;

	Ray ray = d_camPtr->computeCameraRay(i, j, &randState[idx]);
	RayHit hitData, lightHitData;
	Triangle* hitTriPtr;
	Vector3Df hitPt, nextDir, normal, colorAtPixel;
	Vector3Df mask(1.0f, 1.0f, 1.0f);

	// First see if the camera ray hits anything. If not, return black.
	float t = FLT_MAX;
	if (useBvh) {
		t = intersectBVH(d_tris->bvhPtr, d_tris->triPtr, d_tris->bvhIndexPtr, hitData, ray, d_settings->useTexMem);
	}
	else {
		t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, hitData, ray, d_settings->useTexMem);
	}
	if (t < FLT_MAX) {
		hitPt = ray.pointAlong(t);
		hitTriPtr = hitData.hitTriPtr;
		normal = hitTriPtr->getNormal(hitData);
		// if we hit a light directly, add its contribution here so as not to double dip in the BSDF calculations below
		if (hitTriPtr->isEmissive()) {
			d_imgPtr[j * d_settings->width + i] += hitTriPtr->_colorEmit;
			return;
		}
	} else {
		d_imgPtr[j * d_settings->width + i] += Vector3Df(0.0f,0.0f, 0.0f);
		return;
	}


	for (unsigned bounces = 0; bounces < 4; bounces++) {
		// DIFFUSE BSDF:

		// Direct lighting: select light at random, test for intersection, add contribution
		// Get a new ray going towards a random point on the selected light
		float randomNumber = hiprand_uniform(&randState[idx]);
		randomNumber *= (float)d_lights->numLights - 1.0f + 0.9999999f;
		int selectedLightIndex = (int)truncf(randomNumber);
		Triangle selectedLight = d_lights->lightsPtr[selectedLightIndex];
		Vector3Df lightRayDir = normalize(selectedLight.getRandomPointOn(&randState[idx]) - hitPt);

		Ray lightRay(hitPt + normal * EPSILON, lightRayDir);
		if (useBvh){
			t = intersectBVH(d_tris->bvhPtr, d_tris->triPtr, d_tris->bvhIndexPtr, lightHitData, lightRay, d_settings->useTexMem);
		} else {
			t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, lightHitData, lightRay, d_settings->useTexMem);
		}
		if (t < FLT_MAX){
			// See if we've hit the light we tested for
			Triangle* lightRayHitPtr = lightHitData.hitTriPtr;
			if (lightRayHitPtr->_triId == selectedLight._triId) {
				float surfaceArea = selectedLight._surfaceArea;
				float distanceSquared = t*t; // scale by factor of 10
				float incidenceAngle = fabs(dot(selectedLight.getNormal(lightHitData), -lightRayDir));
				float weightFactor = surfaceArea/distanceSquared * incidenceAngle;
				colorAtPixel += mask * selectedLight._colorEmit * hitData.hitTriPtr->_colorDiffuse * weightFactor;
			}
		}

		// Now compute indirect lighting
		if (useBvh) {
			t = intersectBVH(d_tris->bvhPtr, d_tris->triPtr, d_tris->bvhIndexPtr, hitData, ray, d_settings->useTexMem);
		} else {
			t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, hitData, ray, d_settings->useTexMem);
		}
		if (t < FLT_MAX) {

			Vector3Df hitPt = ray.pointAlong(t);
			Triangle* hitTriPtr = hitData.hitTriPtr;
			Vector3Df normal = hitTriPtr->getNormal(hitData);

			if (hitTriPtr->isDiffuse()) {
				float r1 = 2 * M_PI * hiprand_uniform(&randState[idx]);
				float r2 = hiprand_uniform(&randState[idx]);
				float r2sq = sqrtf(r2);

				// calculate orthonormal coordinates u, v, w, at hitpt
				Vector3Df w = normal;
				Vector3Df u = normalize(cross( (fabs(w.x) > 0.1f ?
							Vector3Df(0.f, 1.f, 0.f) :
							Vector3Df(1.f, 0.f, 0.f)), w));
				Vector3Df v = cross(w, u);

				// Random point on unit hemisphere @ hit_point and centered at normal
				nextDir = normalize(u * cosf(r1) * r2sq + v * sinf(r1) * r2sq + w * sqrtf(1.f - r2));
				// Division by 1/2 for this PDF weighted by cosine
				mask *= hitTriPtr->_colorDiffuse * dot(nextDir, normal) * 2.f;
				// Shift hitpoint outward by an epsilon
				hitPt += normal * EPSILON;
			}
			ray = Ray(hitPt, nextDir);
		}
	}
	d_imgPtr[j * d_settings->width + i] += colorAtPixel;
}

__global__ void averageSamplesAndGammaCorrectKernel(Vector3Df* d_streamImgDataPtr, Vector3Df* d_imgPtr, SettingsData* d_settings) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;

	int pixelsInImage = d_settings->width*d_settings->height;
	Vector3Df pixel(0.0f, 0.0f, 0.0f);
	float gamma = 2.2f;
	float invGamma = 1.0f/gamma;
	float invSamples = 1.0f/(float)d_settings->samples;
	for (int s = 0; s < d_settings->numStreams; s++) {
		pixel += d_streamImgDataPtr[idx + s*pixelsInImage];
	}
	pixel *= invSamples;
	d_imgPtr[idx].x = powf(fminf(pixel.x, 1.0f), invGamma);
	d_imgPtr[idx].y = powf(fminf(pixel.y, 1.0f), invGamma);
	d_imgPtr[idx].z = powf(fminf(pixel.z, 1.0f), invGamma);
}

__global__ void setupCurandKernel(hiprandState *randState, int streamOffset) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234 + streamOffset, idx, 0, &randState[idx]);
}

// TODO: Change this to return a bool and keep t in hitData
__device__ float intersectBVH(CacheFriendlyBVHNode* d_bvh,
							  Triangle* d_triPtr,
							  unsigned* d_bvhIndexPtr,
							  RayHit& hitData,
							  const Ray& ray,
							  bool useTexMemory) {
	int stack[BVH_STACK_SIZE];
	int stackIdx = 0;
	stack[stackIdx++] = 0;

	float u, v;
	float t = FLT_MAX;
	float tprime = FLT_MAX;
	// while the stack is not empty
	while (stackIdx) {
		// pop a BVH node from the stack

		int boxIdx = d_bvhIndexPtr[stack[--stackIdx]];
		CacheFriendlyBVHNode* pCurrent = &d_bvh[boxIdx];

		unsigned count = pCurrent->u.leaf._count & 0x7fffffff ;
		if (!(pCurrent->u.leaf._count & 0x80000000)) {   // INNER NODE
			// if ray intersects inner node, push indices of left and right child nodes on the stack
			if (rayIntersectsBox(ray, pCurrent)) {
				stack[stackIdx++] = pCurrent->u.inner._idxRight;
				stack[stackIdx++] = pCurrent->u.inner._idxLeft;
				// return if stack size is exceeded
				if (stackIdx>BVH_STACK_SIZE) {
					printf("Stack size exceeded!\n");
					return FLT_MAX;
				}
			}
		}
		else { // LEAF NODE
			unsigned offset = pCurrent->u.leaf._startIndexInTriIndexList;
			for(int i = 0; i < count; i++){
//				if (i + offset >= 36) {
//					break;
//				}
				tprime = d_triPtr[i + offset].intersect(ray, u, v);
				if (tprime < t && tprime > 0.0f) {
					t = tprime;
					hitData.u = u;
					hitData.v = v;
					hitData.hitTriPtr = &d_triPtr[i + offset];
				}
			}
		}
	}
	return t;
}

__device__ bool rayIntersectsBox(const Ray& ray, CacheFriendlyBVHNode *bvhNode) {
	float t0 = 0.0f, t1 = FLT_MAX;
	float2 bounds;
//	return true;
	// For each axis plane, store bounds and process separately

	// X
	bounds.x = bvhNode->_bottom.x;
	bounds.y = bvhNode->_top.x;
	float invRayDir = 1.f/ray.dir.x;
	float tNear = (bounds.x - ray.origin.x) * invRayDir;
	float tFar = (bounds.y - ray.origin.x) * invRayDir;
	if (tNear > tFar) {
		float tmp = tNear;
		tNear = tFar;
		tFar = tmp;
	}
	t0 = tNear > t0 ? tNear : t0;
	t1 = tFar < t1 ? tFar : t1;
	if (t0 > t1) return false;

	// Y
	bounds.x = bvhNode->_bottom.y;
	bounds.y = bvhNode->_top.y;

	invRayDir = 1.f/ray.dir.y;
	tNear = (bounds.x - ray.origin.y) * invRayDir;
	tFar = (bounds.y - ray.origin.y) * invRayDir;
	if (tNear > tFar) {
		float tmp = tNear;
		tNear = tFar;
		tFar = tmp;
	}
	t0 = tNear > t0 ? tNear : t0;
	t1 = tFar < t1 ? tFar : t1;
	if (t0 > t1) return 0;

	// Z
	bounds.x = bvhNode->_bottom.z;
	bounds.y = bvhNode->_top.z;


	invRayDir = 1.f/ray.dir.z;
	tNear = (bounds.x - ray.origin.z) * invRayDir;
	tFar = (bounds.y - ray.origin.z) * invRayDir;
	if (tNear > tFar) {
		float tmp = tNear;
		tNear = tFar;
		tFar = tmp;
	}
	t0 = tNear > t0 ? tNear : t0;
	t1 = tFar < t1 ? tFar : t1;
	if (t0 > t1) return 0;

	return true;
}

__device__ float intersectTriangles(geom::Triangle* d_triPtr,
									int numTriangles,
									RayHit& hitData,
									const Ray& ray,
									bool useTexMemory) {
	float t = FLT_MAX, tprime = FLT_MAX;
	float u, v;
	for (unsigned i = 0; i < numTriangles; i++) {
		Triangle tri;
		if (useTexMemory) {
			Triangle tri = getTriangleFromTexture(i);
			tprime = tri.intersect(ray, u, v);
		} else {
			tprime = d_triPtr[i].intersect(ray, u, v);
		}
		if (tprime < t && tprime > 0.f) {
			t = tprime;
			hitData.hitTriPtr = &d_triPtr[i];
			hitData.u = u;
			hitData.v = v;
		}
	}
	return t;
}

__device__ inline Triangle getTriangleFromTexture(unsigned i) {
	float4 v1, e1, e2, n1, n2, n3, diff, spec, emit;
	v1 = tex1Dfetch(triangleTexture, i * 9);
	e1 = tex1Dfetch(triangleTexture, i * 9 + 1);
	e2 = tex1Dfetch(triangleTexture, i * 9 + 2);
	n1 = tex1Dfetch(triangleTexture, i * 9 + 3);
	n2 = tex1Dfetch(triangleTexture, i * 9 + 4);
	n3 = tex1Dfetch(triangleTexture, i * 9 + 5);
	diff = tex1Dfetch(triangleTexture, i * 9 + 6);
	spec = tex1Dfetch(triangleTexture, i * 9 + 7);
	emit = tex1Dfetch(triangleTexture, i * 9 + 8);
	return Triangle(v1, e1, e2, n1, n2, n3, diff, spec, emit);
}
