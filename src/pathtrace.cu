#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : pathtrace.cu
 Author      : Tudor Matei Boran
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA raytracer
 ============================================================================
 */
#include "camera.cuh"
#include "cuda_textures.cuh"
#include "pathtrace.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "cuda_error_check.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h
// Random  number generation with CUDA

using namespace geom;

__global__ void debugRenderKernel(Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMem);
__global__ void setupCurandKernel(hiprandState *randState);
__global__ void renderKernel(Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMem, hiprandState *randState);
__global__ void averageSamplesKernel(Vector3Df* d_imgPtr, int width, int height,
		unsigned samples);
__device__ float intersectTriangles(Triangle* d_triPtr, int numTriangles,
		RayHit& hitData, const Ray& ray, bool useTexMem);
__device__ inline Triangle getTriangleFromTexture(unsigned i);

texture_t triangleTexture;

Vector3Df* pathtraceWrapper(Scene& scene, int width, int height, int samples,
		bool &useTexMemory) {
	int pixels = width * height;
	unsigned numTris = scene.getNumTriangles();
	size_t triangleBytes = sizeof(Triangle) * numTris;
	size_t imageBytes = sizeof(Vector3Df) * width * height;

	// Initialize CUDA memory
	Triangle* triPtr = scene.getTriPtr();
	Triangle* d_triPtr = NULL;
	Vector3Df* imgDataPtr = new Vector3Df[pixels];
	Vector3Df* d_imgDataPtr = NULL;
	Camera* camPtr = scene.getCameraPtr();
	Camera* d_camPtr = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_triPtr, triangleBytes));
	CUDA_CHECK_RETURN(
			hipMemcpy((void* )d_triPtr, (void* )triPtr, triangleBytes,
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_imgDataPtr, imageBytes));
	CUDA_CHECK_RETURN(
			hipMemcpy((void* )d_imgDataPtr, (void* )imgDataPtr, imageBytes,
					hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(
			hipMemcpy((void* )d_camPtr, (void* )camPtr, sizeof(Camera),
					hipMemcpyHostToDevice));

	// Bind triangles to texture memory -- texture memory doesn't quite work
	hipArray* d_triDataArray = NULL;
	if (useTexMemory && numTris > TEX_ARRAY_MAX) {
		std::cout << "Not using texture memory because we cannot fit "
				<< numTris << " triangles in 1D hipArray" << std::endl;
		useTexMemory = false;
	}
	if (useTexMemory) {
		std::cout << "Using texture memory!" << std::endl;
		configureTexture(triangleTexture);
		d_triDataArray = bindTrianglesToTexture(triPtr, numTris,
				triangleTexture);
	}

	// Launch kernels
	const unsigned int threadsPerBlock = blockWidth * blockWidth;
	const unsigned int gridBlocks = width / blockWidth * height / blockWidth;
	dim3 block(blockWidth, blockWidth, 1);
	dim3 grid(width / blockWidth, height / blockWidth, 1);

	// Setup cuRand
	hiprandState* d_curandState;
	CUDA_CHECK_RETURN(
			hipMalloc((void** )&d_curandState,
					threadsPerBlock * gridBlocks * sizeof(hiprandState)));
	setupCurandKernel<<<grid, block>>>(d_curandState);

	for (int s = 0; s < samples; s++) {
		renderKernel<<<grid, block>>>(d_triPtr, numTris, d_camPtr, d_imgDataPtr,
				width, height, useTexMemory, d_curandState);
	}

	averageSamplesKernel<<<grid, block>>>(d_imgDataPtr, width, height, samples);

	CUDA_CHECK_RETURN(
			hipMemcpy((void* )imgDataPtr, (void* )d_imgDataPtr, imageBytes,
					hipMemcpyDeviceToHost));
	hipFree((void*) d_triPtr);
	hipFree((void*) d_imgDataPtr);
	hipFree((void*) d_curandState);
	if (useTexMemory)
		hipFreeArray(d_triDataArray);
	return imgDataPtr;
}

__global__ void renderKernel(Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMemory, hiprandState *randState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	Ray ray = d_camPtr->computeCameraRay(i, j, &randState[idx]);
	RayHit hitData;
	Vector3Df colorAtPixel;
	Vector3Df nextDir;
	Vector3Df mask(1.0f, 1.0f, 1.0f);

	for (unsigned bounces = 0; bounces < 6; bounces++) {
		float t = intersectTriangles(d_triPtr, numTriangles, hitData, ray,
				useTexMemory);
		if (t < MAX_DISTANCE) {
			Vector3Df hitPt = ray.pointAlong(t);
			Triangle* hitTriPtr = hitData.hitTriPtr;
			Vector3Df normal = hitTriPtr->getNormal(hitData);
			colorAtPixel += mask * hitTriPtr->_colorEmit;

			if (hitTriPtr->isDiffuse()) {
				float r1 = 2 * M_PI * hiprand_uniform(&randState[idx]);
				float r2 = hiprand_uniform(&randState[idx]);
				float r2sq = sqrtf(r2);

				// calculate orthonormal coordinates u, v, w, at hitpt
				Vector3Df w = normal;
				Vector3Df u = normalize(
						cross(
								(fabs(w.x) > 0.1f ?
										make_float3(0.f, 1.f, 0.f) :
										make_float3(1.f, 0.f, 0.f)), w));
				Vector3Df v = cross(w, u);

				// Random point on unit hemisphere @ hit_point and centered at normal
				nextDir = normalize(
						u * cosf(r1) * r2sq + v * sinf(r1) * r2sq
								+ w * sqrtf(1.f - r2));
				// Division by 1/2 for this PDF weighted by cosine
				mask *= hitTriPtr->_colorDiffuse * dot(nextDir, normal) * 2.f;
				// Shift hitpoint outward by an epsilon
				hitPt += normal * EPSILON;
			}
			ray = Ray(hitPt, nextDir);
		}
	}
	d_imgPtr[j * width + i] += colorAtPixel;
}

__global__ void debugRenderKernel(geom::Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMemory) {
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	hiprandState d_curandState;
	hiprand_init(1234, i * j + i, 0, &d_curandState);

	Ray camRay = d_camPtr->computeCameraRay(i, j, &d_curandState);
	RayHit hitData;
	float t = intersectTriangles(d_triPtr, numTriangles, hitData, camRay,
			useTexMemory);
	Vector3Df light(0.0f, 10.0f, 1.0f);
	if (t < MAX_DISTANCE) {
		Vector3Df hitPt = camRay.pointAlong(t);
		Vector3Df lightDir = normalize(light - hitPt);
		Vector3Df normal = hitData.hitTriPtr->getNormal(hitData);
		d_imgPtr[j * width + i] = Vector3Df(
				hitData.hitTriPtr->_colorDiffuse
						* max(dot(lightDir, normal), 0.0f));
	}
}

__global__ void averageSamplesKernel(Vector3Df* d_imgPtr, int width, int height,
		unsigned samples) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	d_imgPtr[idx] *= 1.0f / (float) samples;
	d_imgPtr[idx].x = fminf(d_imgPtr[idx].x, 1.0f);
	d_imgPtr[idx].y = fminf(d_imgPtr[idx].y, 1.0f);
	d_imgPtr[idx].z = fminf(d_imgPtr[idx].z, 1.0f);
}

__global__ void setupCurandKernel(hiprandState *randState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &randState[idx]);
}

__device__ float intersectTriangles(geom::Triangle* d_triPtr, int numTriangles,
		RayHit& hitData, const Ray& ray, bool useTexMemory) {
	float t = MAX_DISTANCE, tprime = MAX_DISTANCE;
	float u, v;
	for (unsigned i = 0; i < numTriangles; i++) {
		Triangle tri;
		if (useTexMemory) {
			Triangle tri = getTriangleFromTexture(i);
			tprime = tri.intersect(ray, u, v);
		} else {
			tprime = d_triPtr[i].intersect(ray, u, v);
		}
		if (tprime < t && tprime > 0.f) {
			t = tprime;
			hitData.hitTriPtr = &d_triPtr[i];
			hitData.u = u;
			hitData.v = v;
		}
	}
	return t;
}

__device__ inline Triangle getTriangleFromTexture(unsigned i) {
	float4 v1, e1, e2, n1, n2, n3, diff, spec, emit;
	v1 = tex1Dfetch(triangleTexture, i * 9);
	e1 = tex1Dfetch(triangleTexture, i * 9 + 1);
	e2 = tex1Dfetch(triangleTexture, i * 9 + 2);
	n1 = tex1Dfetch(triangleTexture, i * 9 + 3);
	n2 = tex1Dfetch(triangleTexture, i * 9 + 4);
	n3 = tex1Dfetch(triangleTexture, i * 9 + 5);
	diff = tex1Dfetch(triangleTexture, i * 9 + 6);
	spec = tex1Dfetch(triangleTexture, i * 9 + 7);
	emit = tex1Dfetch(triangleTexture, i * 9 + 8);
	return Triangle(v1, e1, e2, n1, n2, n3, diff, spec, emit);
}

