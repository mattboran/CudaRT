#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : pathtrace.cu
 Author      : Tudor Matei Boran
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA raytracer
 ============================================================================
 */
#include "camera.cuh"
#include "cuda_textures.cuh"
#include "pathtrace.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "cuda_error_check.h" // includes hip/hip_runtime.h and hip/hip_runtime_api.h

using namespace geom;

struct LightsData {
	Triangle* lightsPtr;
	unsigned numLights;
	float totalSurfaceArea;
};

struct TrianglesData {
	Triangle* triPtr;
	unsigned numTriangles;
};

// TODO: Move image, camera, and hiprandState pointers into here
struct SettingsData {
	int width;
	int height;
	int samples;
	bool useTexMem;
};

__global__ void debugRenderKernel(Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMem);
__global__ void setupCurandKernel(hiprandState *randState);
__global__ void renderKernel(TrianglesData* d_tris, Camera* d_camPtr, Vector3Df* d_imgPtr, LightsData* d_lights, SettingsData* d_settings, hiprandState *randState);
__global__ void averageSamplesKernel(Vector3Df* d_imgPtr, SettingsData* d_settings);
__device__ float intersectTriangles(Triangle* d_triPtr, int numTriangles, RayHit& hitData, const Ray& ray, bool useTexMem);
__device__ inline Triangle getTriangleFromTexture(unsigned i);


texture_t triangleTexture;

Vector3Df* pathtraceWrapper(Scene& scene, int width, int height, int samples, bool &useTexMemory) {
	int pixels = width * height;
	unsigned numTris = scene.getNumTriangles();
	size_t triangleBytes = sizeof(Triangle) * numTris;
	size_t imageBytes = sizeof(Vector3Df) * width * height;

	// Initialize CUDA memory

	// Triangles -> d_tris
	Triangle* h_triPtr = scene.getTriPtr();
	Triangle* d_triPtr = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_triPtr, triangleBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_triPtr, (void* )h_triPtr, triangleBytes, hipMemcpyHostToDevice));

	TrianglesData* h_tris = (TrianglesData*)malloc(sizeof(TrianglesData) + triangleBytes);
	TrianglesData* d_tris = NULL;
	h_tris->numTriangles = numTris;
	h_tris->triPtr = d_triPtr;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_tris, sizeof(TrianglesData) + triangleBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_tris, (void*)h_tris, sizeof(TrianglesData) + triangleBytes, hipMemcpyHostToDevice));

	// Lights -> d_lights
	Triangle* lightsPtr = scene.getLightsPtr();
	Triangle* d_lightTrianglePtr = NULL;
	size_t lightTrianglesBytes = sizeof(Triangle) * scene.getNumLights();
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_lightTrianglePtr, lightTrianglesBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_lightTrianglePtr, (void *)lightsPtr, lightTrianglesBytes, hipMemcpyHostToDevice));

	LightsData* h_lights = (LightsData*)malloc(sizeof(LightsData));
	LightsData* d_lights = NULL;
	h_lights->lightsPtr = d_lightTrianglePtr;
	h_lights->numLights = scene.getNumLights();
	h_lights->totalSurfaceArea = scene.getLightsSurfaceArea();
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_lights, sizeof(LightsData) + lightTrianglesBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_lights, h_lights, sizeof(LightsData) + lightTrianglesBytes, hipMemcpyHostToDevice));

	// Setup settings -> d_settings
	SettingsData h_settings;
	SettingsData* d_settings;
	h_settings.width = width;
	h_settings.height = height;
	h_settings.samples = samples;
	h_settings.useTexMem = useTexMemory;
	CUDA_CHECK_RETURN(hipMalloc((void**)&d_settings, sizeof(SettingsData)));
	CUDA_CHECK_RETURN(hipMemcpy((void*)d_settings, &h_settings, sizeof(SettingsData), hipMemcpyHostToDevice));

	// Image
	Vector3Df* imgDataPtr = new Vector3Df[pixels]();
	Vector3Df* d_imgDataPtr = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_imgDataPtr, imageBytes));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_imgDataPtr, (void* )imgDataPtr, imageBytes, hipMemcpyHostToDevice));

	// Camera
	Camera* camPtr = scene.getCameraPtr();
	Camera* d_camPtr = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_camPtr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMemcpy((void* )d_camPtr, (void* )camPtr, sizeof(Camera), hipMemcpyHostToDevice));

	// Bind triangles to texture memory -- texture memory doesn't quite work
	hipArray* d_triDataArray = NULL;
	if (useTexMemory && numTris > TEX_ARRAY_MAX) {
		std::cout << "Not using texture memory because we cannot fit "
				<< numTris << " triangles in 1D hipArray" << std::endl;
		useTexMemory = false;
	}
	if (useTexMemory) {
		std::cout << "Using texture memory!" << std::endl;
		configureTexture(triangleTexture);
		d_triDataArray = bindTrianglesToTexture(h_triPtr, numTris,
				triangleTexture);
	}

	// Launch kernels
	const unsigned int threadsPerBlock = blockWidth * blockWidth;
	const unsigned int gridBlocks = width / blockWidth * height / blockWidth;
	dim3 block(blockWidth, blockWidth, 1);
	dim3 grid(width / blockWidth, height / blockWidth, 1);

	// Setup cuRand kernel
	hiprandState* d_curandState;
	CUDA_CHECK_RETURN(hipMalloc((void** )&d_curandState, threadsPerBlock * gridBlocks * sizeof(hiprandState)));
	setupCurandKernel<<<grid, block>>>(d_curandState);

	for (int s = 0; s < samples; s++) {
		renderKernel<<<grid, block>>>(d_tris, d_camPtr, d_imgDataPtr, d_lights, d_settings, d_curandState);
	}

	averageSamplesKernel<<<grid, block>>>(d_imgDataPtr, d_settings);

	CUDA_CHECK_RETURN(hipMemcpy((void* )imgDataPtr, (void* )d_imgDataPtr, imageBytes, hipMemcpyDeviceToHost));

	free(h_lights);
	free(h_tris);
	hipFree((void*) d_lightTrianglePtr);
	hipFree((void*) d_triPtr);
	hipFree((void*) d_tris);
	hipFree((void*) d_settings);
	hipFree((void*) d_imgDataPtr);
	hipFree((void*) d_curandState);
	if (useTexMemory)
		hipFreeArray(d_triDataArray);
	return imgDataPtr;
}

__global__ void renderKernel(TrianglesData* d_tris,
							Camera* d_camPtr,
							Vector3Df* d_imgPtr,
							LightsData* d_lights,
							SettingsData* d_settings,
							hiprandState *randState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;

	Ray ray = d_camPtr->computeCameraRay(i, j, &randState[idx]);
	RayHit hitData, lightHitData;
	Triangle* hitTriPtr;
	Vector3Df hitPt, normal, nextDir, colorAtPixel;
	Vector3Df mask(1.0f, 1.0f, 1.0f);

	// First see if the camera ray hits anything. If not, return black.
	float t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, hitData, ray, d_settings->useTexMem);
	if (t < MAX_DISTANCE) {
		hitPt = ray.pointAlong(t);
		hitTriPtr = hitData.hitTriPtr;
		normal = hitTriPtr->getNormal(hitData);
	} else {
		d_imgPtr[j * d_settings->width + i] += Vector3Df(0.0f, 0.0f, 0.0f);
		return;
	}

	// Direct lighting: select light at random, test for intersection, add contribution
	// Get a new ray going towards a random point on the selected light
	float randomNumber = hiprand_uniform(&randState[idx]);
	randomNumber *= (float)d_lights->numLights - 1.0f + 0.9999999f;
	int selectedLightIndex = (int)truncf(randomNumber);
	Triangle selectedLight = d_lights->lightsPtr[selectedLightIndex];

	Vector3Df lightRayDir = selectedLight.getPointOn(&randState[idx]) - hitPt;
	Ray lightRay(hitPt + normal * 0.01f, lightRayDir);
	t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, lightHitData, lightRay, d_settings->useTexMem);
	if (t > lightRayDir.length())  {
		float distanceFactor = 1.0f;
		float numLightsFactor = 1.0f/(float)d_lights->numLights;
		colorAtPixel = selectedLight._colorEmit * hitData.hitTriPtr->_colorDiffuse * distanceFactor * numLightsFactor;
	}

	for (unsigned bounces = 0; bounces < 4; bounces++) {
		t = intersectTriangles(d_tris->triPtr, d_tris->numTriangles, hitData, ray, d_settings->useTexMem);
		if (t < MAX_DISTANCE) {

			Vector3Df hitPt = ray.pointAlong(t);
			Triangle* hitTriPtr = hitData.hitTriPtr;
			Vector3Df normal = hitTriPtr->getNormal(hitData);

			colorAtPixel += mask * hitTriPtr->_colorEmit;

			if (hitTriPtr->isDiffuse()) {
				float r1 = 2 * M_PI * hiprand_uniform(&randState[idx]);
				float r2 = hiprand_uniform(&randState[idx]);
				float r2sq = sqrtf(r2);

				// calculate orthonormal coordinates u, v, w, at hitpt
				Vector3Df w = normal;
				Vector3Df u = normalize(cross( (fabs(w.x) > 0.1f ?
							Vector3Df(0.f, 1.f, 0.f) :
							Vector3Df(1.f, 0.f, 0.f)), w));
				Vector3Df v = cross(w, u);

				// Random point on unit hemisphere @ hit_point and centered at normal
				nextDir = normalize(u * cosf(r1) * r2sq + v * sinf(r1) * r2sq + w * sqrtf(1.f - r2));
				// Division by 1/2 for this PDF weighted by cosine
				mask *= hitTriPtr->_colorDiffuse * dot(nextDir, normal) * 2.f;
				// Shift hitpoint outward by an epsilon
				hitPt += normal * EPSILON;
			}
			ray = Ray(hitPt, nextDir);
		}
	}
	d_imgPtr[j * d_settings->width + i] += colorAtPixel;
}

__global__ void debugRenderKernel(geom::Triangle* d_triPtr, int numTriangles,
		Camera* d_camPtr, Vector3Df* d_imgPtr, int width, int height,
		bool useTexMemory) {
	unsigned int i, j;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	hiprandState d_curandState;
	hiprand_init(1234, i * j + i, 0, &d_curandState);

	Ray camRay = d_camPtr->computeCameraRay(i, j, &d_curandState);
	RayHit hitData;
	float t = intersectTriangles(d_triPtr, numTriangles, hitData, camRay,
			useTexMemory);
	Vector3Df light(0.0f, 10.0f, 1.0f);
	if (t < MAX_DISTANCE) {
		Vector3Df hitPt = camRay.pointAlong(t);
		Vector3Df lightDir = normalize(light - hitPt);
		Vector3Df normal = hitData.hitTriPtr->getNormal(hitData);
		d_imgPtr[j * width + i] = Vector3Df(
				hitData.hitTriPtr->_colorDiffuse
						* max(dot(lightDir, normal), 0.0f));
	}
}

__global__ void averageSamplesKernel(Vector3Df* d_imgPtr, SettingsData* d_settings) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	d_imgPtr[idx] *= 1.0f / (float) d_settings->samples;
	d_imgPtr[idx].x = fminf(d_imgPtr[idx].x, 1.0f);
	d_imgPtr[idx].y = fminf(d_imgPtr[idx].y, 1.0f);
	d_imgPtr[idx].z = fminf(d_imgPtr[idx].z, 1.0f);
}

__global__ void setupCurandKernel(hiprandState *randState) {
	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y)
			+ (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprand_init(1234, idx, 0, &randState[idx]);
}

__device__ float intersectTriangles(geom::Triangle* d_triPtr,
									int numTriangles,
									RayHit& hitData,
									const Ray& ray,
									bool useTexMemory) {
	float t = MAX_DISTANCE, tprime = MAX_DISTANCE;
	float u, v;
	for (unsigned i = 0; i < numTriangles; i++) {
		Triangle tri;
		if (useTexMemory) {
			Triangle tri = getTriangleFromTexture(i);
			tprime = tri.intersect(ray, u, v);
		} else {
			tprime = d_triPtr[i].intersect(ray, u, v);
		}
		if (tprime < t && tprime > 0.f) {
			t = tprime;
			hitData.hitTriPtr = &d_triPtr[i];
			hitData.u = u;
			hitData.v = v;
		}
	}
	return t;
}

__device__ inline Triangle getTriangleFromTexture(unsigned i) {
	float4 v1, e1, e2, n1, n2, n3, diff, spec, emit;
	v1 = tex1Dfetch(triangleTexture, i * 9);
	e1 = tex1Dfetch(triangleTexture, i * 9 + 1);
	e2 = tex1Dfetch(triangleTexture, i * 9 + 2);
	n1 = tex1Dfetch(triangleTexture, i * 9 + 3);
	n2 = tex1Dfetch(triangleTexture, i * 9 + 4);
	n3 = tex1Dfetch(triangleTexture, i * 9 + 5);
	diff = tex1Dfetch(triangleTexture, i * 9 + 6);
	spec = tex1Dfetch(triangleTexture, i * 9 + 7);
	emit = tex1Dfetch(triangleTexture, i * 9 + 8);
	return Triangle(v1, e1, e2, n1, n2, n3, diff, spec, emit);
}

